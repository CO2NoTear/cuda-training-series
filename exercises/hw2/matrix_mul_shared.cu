
#include <hip/hip_runtime.h>
#include <stdio.h>

// these are just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg)                                                   \
  do {                                                                         \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,                  \
              hipGetErrorString(__err), __FILE__, __LINE__);                  \
      fprintf(stderr, "*** FAILED - ABORTING\n");                              \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

const int DSIZE = 8192;
const int block_size = 32; // CUDA maximum is 1024 *total* threads in block
const float A_val = 3.0f;
const float B_val = 2.0f;

// matrix multiply (naive) kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int idx = threadIdx.x + blockDim.x * blockIdx.x; // create thread x index
  int idy = threadIdx.y + blockDim.y * blockIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds)) {
    float temp = 0;
    for (int i = 0; i < ds / block_size; i++) {

      // Load data into shared memory
      // 这里的i是在8192/32 = 256个block间迭代，block内部的threadID不变
      As[threadIdx.y][threadIdx.x] =
          A[ds * idy + i * block_size + threadIdx.x]; // A for row
      Bs[threadIdx.y][threadIdx.x] =
          B[ds * (i * block_size + threadIdx.y) + idx]; // B for col

      // Synchronize
      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
        temp += As[k][threadIdx.x] *
                Bs[threadIdx.y][k]; // dot product of row and column
      __syncthreads();
    }

    // Write to global memory
    C[idy * ds + idx] = temp;
  }
}

int main() {

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

  // these are just for timing
  clock_t t0, t1, t2;
  double t1sum = 0.0;
  double t2sum = 0.0;

  // start timing
  t0 = clock();

  h_A = new float[DSIZE * DSIZE];
  h_B = new float[DSIZE * DSIZE];
  h_C = new float[DSIZE * DSIZE];
  for (int i = 0; i < DSIZE * DSIZE; i++) {
    h_A[i] = A_val;
    h_B[i] = B_val;
    h_C[i] = 0;
  }

  // Initialization timing
  t1 = clock();
  t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
  printf("Init took %f seconds.  Begin compute\n", t1sum);

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
  hipMalloc(&d_B, DSIZE * DSIZE * sizeof(float));
  hipMalloc(&d_C, DSIZE * DSIZE * sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel
  dim3 block(block_size, block_size); // dim3 variable holds 3 dimensions
  dim3 grid((DSIZE + block.x - 1) / block.x, (DSIZE + block.y - 1) / block.y);
  mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete

  // Copy results back to host
  hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(float), hipMemcpyDeviceToHost);

  // GPU timing
  t2 = clock();
  t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
  printf("Done. Compute took %f seconds\n", t2sum);

  // Cuda processing sequence step 3 is complete

  // Verify results
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < DSIZE * DSIZE; i++)
    if (h_C[i] != A_val * B_val * DSIZE) {
      printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i],
             A_val * B_val * DSIZE);
      return -1;
    }
  printf("Success!\n");
  return 0;
}
